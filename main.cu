
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

typedef struct mp
{
  float x, y; // translation vertex
  float a, b, c, d; // scaling/rotation matrix
  float p; // mapping probability
} mapping;

__global__ void kernel(float4* d_pointData, int numPoints, mapping *d_mappings, int numMappings)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  // If needed for performance, move curand_init to seperate kernel and store
  // states in device memory
  hiprandState state;
  hiprand_init((unsigned long long) clock(), index, 0, &state);

  // Set up transformation mapping once per block in shared memory
  extern __shared__ mapping maps[];
  if(threadIdx.x == 0)
  {
    for(int i = 0; i < numMappings; i++)
        maps[i] = d_mappings[i];
  }
  __syncthreads();

  // Initially start at a mapping vertex to guarantee we stay inside the
  // iterated function system
  int currentTarget = index % numMappings;
  float2 currentPosition, newPosition;
  currentPosition.x = maps[currentTarget].x;
  currentPosition.y = maps[currentTarget].y;

  for(int i = index; i < numPoints; i += stride)
  {
    // set the current vertex to the currentPosition
    d_pointData[i].x = currentPosition.x;
    d_pointData[i].y = currentPosition.y;

    // set the iteration percentage and current target mapping
    d_pointData[i].z =  i / (float) numPoints;
    d_pointData[i].w = currentTarget;

    // find random target with given mapping probabilities
    // If needed for performance, find method to remove thread divergence
    // Note: changing 4 to numMappings in for loop reduced performance 50%
    float currentProb = hiprand_uniform(&state);
    float totalProb = 0.0f;
    for(int j = 0; j < numMappings; j++)
    {
        totalProb += maps[j].p;
        if(currentProb < totalProb)
        {
            currentTarget = j;
            break;
        }
    }

    // calculate the transformation
    // (x_n+1) = (a b)(x_n) + (e)
    // (y_n+1)   (c d)(y_n)   (f)
    newPosition.x = maps[currentTarget].a * currentPosition.x +
                    maps[currentTarget].b * currentPosition.y +
                    maps[currentTarget].x;
    newPosition.y = maps[currentTarget].c * currentPosition.x +
                    maps[currentTarget].d * currentPosition.y +
                    maps[currentTarget].y;
    currentPosition = newPosition;
  }
}

int main() {
	const int numBlocks = 1;
	const int blockSize = 1024;
	const int numMappings = 0;
	const int numPoints = 1000000;

	mapping *d_map;
	float4 *d_glmap;
	hipMalloc(&d_map, sizeof(mapping) * numMappings);
	hipMalloc(&d_glmap, sizeof(float4) * numPoints);


	kernel<<<numBlocks, blockSize, numMappings * sizeof(mapping)>>>
      ((float4*)d_glmap, numPoints, d_map, numMappings);

	hipFree(d_map);
	hipFree(d_glmap);
}
